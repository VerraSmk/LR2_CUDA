#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>

void reductionWithCudaImproved(float* result, const float* input);
__global__ void reductionKernelImproved(float* result, const float* input);
void reductionCPU(float* result, const float* input);

#define SIZE 10000000
#define TILE 32
#define ILP 8
#define BLOCK_X_IMPR (TILE / ILP)
#define BLOCK_Y_IMPR 32
#define BLOCK_COUNT_X_IMPR 100


void reductionCPU(float* result, const float* input)
{
    for (int i = 0; i < SIZE; i++)
        *result += input[i];
}

__global__ void reductionKernelImproved(float* result, const float* input)
{
    int i;
    int col = (blockDim.x * blockIdx.x + threadIdx.x) * ILP;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int index = row * blockDim.x * gridDim.x * ILP + col;
    __shared__ float interResult;

    if (threadIdx.x == 0 && threadIdx.y == 0)
        interResult = 0.0;

    __syncthreads();

#pragma unroll ILP
    for (i = 0; i < ILP; i++)
    {
        if (index < SIZE)
        {
            atomicAdd(&interResult, input[index]);
            index++;
        }
    }

    __syncthreads();

    if (threadIdx.x == 0 && threadIdx.y == 0)
        atomicAdd(result, interResult);
}

void reductionWithCudaImproved(float* result, const float* input)
{
    dim3 dim_grid, dim_block;

    float* dev_input = 0;
    float* dev_result = 0;
    hipEvent_t start, stop;
    float elapsed = 0;
    double gpuBandwidth;

    dim_block.x = BLOCK_X_IMPR;
    dim_block.y = BLOCK_Y_IMPR;
    dim_block.z = 1;

    dim_grid.x = BLOCK_COUNT_X_IMPR;
    dim_grid.y = (int)ceil((float)SIZE / (float)(TILE * dim_block.y * BLOCK_COUNT_X_IMPR));
    dim_grid.z = 1;

    hipSetDevice(0);

    hipMalloc((void**)&dev_input, SIZE * sizeof(float));
    hipMalloc((void**)&dev_result, sizeof(float));
    hipMemcpy(dev_input, input, SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_result, result, sizeof(float), hipMemcpyHostToDevice);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    reductionKernelImproved << <dim_grid, dim_block >> > (dev_result, dev_input);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsed, start, stop);

    printf("GPU: %f ms\n", elapsed);

    hipDeviceSynchronize();

    hipMemcpy(result, dev_result, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_input);
    hipFree(dev_result);

    return;
}


int main()
{
    int i;
    float* input;
    float resultCPU, resultGPU;
    double cpuTime, cpuBandwidth;
    printf("Size : %d \n", SIZE);
    input = (float*)malloc(SIZE * sizeof(float));
    resultCPU = 0.0;
    resultGPU = 0.0;

    srand((int)time(NULL));

    auto start = std::chrono::high_resolution_clock::now();
    auto end = std::chrono::high_resolution_clock::now();

    for (i = 0; i < SIZE; i++)
        input[i] = rand() % 10 - 5;

    start = std::chrono::high_resolution_clock::now();
    reductionCPU(&resultCPU, input);
    end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> diff = end - start;
    cpuTime = (diff.count() * 1000);
    printf("CPU: %f ms\n", cpuTime);

    reductionWithCudaImproved(&resultGPU, input);

    return 0;
}